#include <bits/stdc++.h>
#include <chrono>

#include <hip/hip_runtime.h>

#define BLOCK 16 // 各ブロックは16 x 16個のスレッドから定義されるものとする．
#define WIDTH 1024 // 処理対象の行列のサイズはWIDTH x WIDTH.

// ホスト(CPU)側の行列定義．                   
float h_A[WIDTH * WIDTH];
float h_B[WIDTH * WIDTH];
float h_C[WIDTH * WIDTH];

// デバイス(GPU)側の行列へのポインタ．
float* d_A, * d_B, * d_C;

void h_multiply(float* A, float* B, float* C);
__global__ void d_multiply0(float* A, float* B, float* C);
__global__ void d_multiply1(float* A, float* B, float* C); 

// メイン関数．
int main()
{
	unsigned int i;

	// デバイス側に行列用のメモリを確保．
	hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);

	// ホスト側の行列に値をセット．
	for (i = 0; i < (WIDTH * WIDTH); i++)
	{
		h_A[i] = (float)i;
		h_B[i] = (float)i;
	}

	// 計算時間の計測開始．
	auto d_start = std::chrono::system_clock::now();

	// ホスト側の行列のデータをデバイス側の行列へ転送．
	hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);

	// グリッドとブロックの定義．
	dim3 grid(WIDTH / BLOCK, WIDTH / BLOCK);
	dim3 block(BLOCK, BLOCK);

	// GPU処理の起動．
	d_multiply0 <<< grid, block >>> (d_A, d_B, d_C);

	// 計算結果はd_cに格納されているので，それをホスト側のh_Cに転送．
	hipMemcpy(h_C, d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

	// 計算時間の計測終了．
	auto d_end = std::chrono::system_clock::now();
	auto d_calcTime = d_end - d_start;

	// 計算結果の表示．
	std::cout << "デバイス計算時間: " << std::chrono::duration_cast<std::chrono::milliseconds>(d_calcTime).count() << "(ms) ";
	std::cout << "デバイス計算結果: " << h_C[WIDTH * WIDTH - 1] << std::endl;

	// デバイス側のメモリを解放．
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// 比較用にホスト側でも計算してみる．
	auto h_start = std::chrono::system_clock::now();
	h_multiply(h_A, h_B, h_C);
	auto h_end = std::chrono::system_clock::now();
	auto h_calcTime = h_end - h_start;
	std::cout << " ホスト計算時間: " << std::chrono::duration_cast<std::chrono::milliseconds>(h_calcTime).count() << "(ms) ";
	std::cout << " ホスト計算結果: " << h_C[WIDTH * WIDTH - 1] << std::endl;
}

void h_multiply(float* A, float* B, float* C)
{
	unsigned int r, c, i;
	float tmp;
	for (r = 0; r < WIDTH; r++) {
		for (c = 0; c < WIDTH; c++) {
			tmp = 0.0;
			for (i = 0; i < WIDTH; i++)
				tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
			C[WIDTH * r + c] = tmp;
		}
	}
}

__global__ void d_multiply0(float* A, float* B, float* C)
{
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y; // スレッドが担当する行番号．
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x; // スレッドが担当する列番号．
	unsigned int i;
	float tmp;
	tmp = 0.0f;
	for (i = 0; i < WIDTH; i++)
		tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
	C[WIDTH * r + c] = tmp;
}

__global__ void d_multiply1(float* A, float* B, float* C)
{
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int i, j;
	float tmp;
	__shared__ float s_A[BLOCK][BLOCK];
	__shared__ float s_B[BLOCK][BLOCK];
	tmp = 0.0f;
	for (i = 0; i < WIDTH; i++) {

		// 行列の一部をシェアードメモリに確保．
		s_A[threadIdx.y][threadIdx.x] = A[WIDTH * r + i + threadIdx.x];
		s_B[threadIdx.y][threadIdx.y] = A[WIDTH * (i + threadIdx.y) + c];
		__syncthreads();

		// シェアードメモリで積を計算．
		for (j = 0; j < BLOCK; j++)
			tmp += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];
		__syncthreads();
	}
	C[WIDTH * r + c] = tmp;

}