#include <bits/stdc++.h>
#include <chrono>

#include <hip/hip_runtime.h>

#define BLOCK 16 // �e�u���b�N��16 x 16�̃X���b�h�����`�������̂Ƃ���D
#define WIDTH 1024 // �����Ώۂ̍s��̃T�C�Y��WIDTH x WIDTH.

// �z�X�g(CPU)���̍s���`�D                   
float h_A[WIDTH * WIDTH];
float h_B[WIDTH * WIDTH];
float h_C[WIDTH * WIDTH];

// �f�o�C�X(GPU)���̍s��ւ̃|�C���^�D
float* d_A, * d_B, * d_C;

void h_multiply(float* A, float* B, float* C);
__global__ void d_multiply0(float* A, float* B, float* C);
__global__ void d_multiply1(float* A, float* B, float* C); 

// ���C���֐��D
int main()
{
	unsigned int i;

	// �f�o�C�X���ɍs��p�̃��������m�ہD
	hipMalloc((void**)&d_A, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_B, sizeof(float) * WIDTH * WIDTH);
	hipMalloc((void**)&d_C, sizeof(float) * WIDTH * WIDTH);

	// �z�X�g���̍s��ɒl���Z�b�g�D
	for (i = 0; i < (WIDTH * WIDTH); i++)
	{
		h_A[i] = (float)i;
		h_B[i] = (float)i;
	}

	// �v�Z���Ԃ̌v���J�n�D
	auto d_start = std::chrono::system_clock::now();

	// �z�X�g���̍s��̃f�[�^���f�o�C�X���̍s��֓]���D
	hipMemcpy(d_A, h_A, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, sizeof(float) * WIDTH * WIDTH, hipMemcpyHostToDevice);

	// �O���b�h�ƃu���b�N�̒�`�D
	dim3 grid(WIDTH / BLOCK, WIDTH / BLOCK);
	dim3 block(BLOCK, BLOCK);

	// GPU�����̋N���D
	d_multiply0 <<< grid, block >>> (d_A, d_B, d_C);

	// �v�Z���ʂ�d_c�Ɋi�[����Ă���̂ŁC������z�X�g����h_C�ɓ]���D
	hipMemcpy(h_C, d_C, sizeof(float) * WIDTH * WIDTH, hipMemcpyDeviceToHost);

	// �v�Z���Ԃ̌v���I���D
	auto d_end = std::chrono::system_clock::now();
	auto d_calcTime = d_end - d_start;

	// �v�Z���ʂ̕\���D
	std::cout << "�f�o�C�X�v�Z����: " << std::chrono::duration_cast<std::chrono::milliseconds>(d_calcTime).count() << "(ms) ";
	std::cout << "�f�o�C�X�v�Z����: " << h_C[WIDTH * WIDTH - 1] << std::endl;

	// �f�o�C�X���̃�����������D
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	// ��r�p�Ƀz�X�g���ł��v�Z���Ă݂�D
	auto h_start = std::chrono::system_clock::now();
	h_multiply(h_A, h_B, h_C);
	auto h_end = std::chrono::system_clock::now();
	auto h_calcTime = h_end - h_start;
	std::cout << " �z�X�g�v�Z����: " << std::chrono::duration_cast<std::chrono::milliseconds>(h_calcTime).count() << "(ms) ";
	std::cout << " �z�X�g�v�Z����: " << h_C[WIDTH * WIDTH - 1] << std::endl;
}

void h_multiply(float* A, float* B, float* C)
{
	unsigned int r, c, i;
	float tmp;
	for (r = 0; r < WIDTH; r++) {
		for (c = 0; c < WIDTH; c++) {
			tmp = 0.0;
			for (i = 0; i < WIDTH; i++)
				tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
			C[WIDTH * r + c] = tmp;
		}
	}
}

__global__ void d_multiply0(float* A, float* B, float* C)
{
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y; // �X���b�h���S������s�ԍ��D
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x; // �X���b�h���S�������ԍ��D
	unsigned int i;
	float tmp;
	tmp = 0.0f;
	for (i = 0; i < WIDTH; i++)
		tmp += A[WIDTH * r + i] * B[WIDTH * i + c];
	C[WIDTH * r + c] = tmp;
}

__global__ void d_multiply1(float* A, float* B, float* C)
{
	unsigned int r = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int c = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int i, j;
	float tmp;
	__shared__ float s_A[BLOCK][BLOCK];
	__shared__ float s_B[BLOCK][BLOCK];
	tmp = 0.0f;
	for (i = 0; i < WIDTH; i++) {

		// �s��̈ꕔ���V�F�A�[�h�������Ɋm�ہD
		s_A[threadIdx.y][threadIdx.x] = A[WIDTH * r + i + threadIdx.x];
		s_B[threadIdx.y][threadIdx.y] = A[WIDTH * (i + threadIdx.y) + c];
		__syncthreads();

		// �V�F�A�[�h�������Őς��v�Z�D
		for (j = 0; j < BLOCK; j++)
			tmp += s_A[threadIdx.y][j] * s_B[j][threadIdx.x];
		__syncthreads();
	}
	C[WIDTH * r + c] = tmp;

}